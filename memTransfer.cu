#include<hip/hip_runtime.h>
#include<stdio.h>

int main(int argc, char **argv){
    // set up device
    int dev = 0;
    hipSetDevice(dev);

    // memory size
    unsigned int isize = 1 << 22;
    unsigned int nbytes = isize * sizeof(float); 

    // get device information
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting at", argv[0]); 
    printf("device %d: %s memory size %d nbyte %5.2fMB\n",dev,
        deviceProp.name, isize, nbytes/(1024.0f * 1024.0f));
    
    // allocate the host memory
    float *h_a = (float *)malloc(nbytes);

    // allocate the device memory
    float *d_a;
    hipMalloc((float **)&d_a, nbytes);

    // initialize the host memory
    memset(h_a, 0.5f, nbytes);

    // transfer data from the host to the device
    hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);

    // free memory
    hipFree(d_a);
    free(h_a);

    // reset device
    hipDeviceReset();
    return EXIT_SUCCESS;

}